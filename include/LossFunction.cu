#include "hip/hip_runtime.h"
#ifndef LOSS_FUNCTION
#define LOSS_FUNCTION

#include"CalculateNodeBase.cu"
#include"Tensor.cu"

template<class T>
class MSELoss:public CalculateNodeBase<T>{
private:
    bool m_average = true;
    bool m_reduce = true;

public:
    
    MSELoss(bool average = true , bool reduce = true):m_average(average) , m_reduce(reduce) {};

    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        if(data.size() != 2) return nullptr;
        auto input = *(data.begin());
        auto label = *(data.begin() + 1);
        input->addUseTime();
        label->addUseTime();
        preTensorNodes["input"] = input;
        preTensorNodes["label"] = label;
        if(!m_reduce){
            if(!backTensorNode || backTensorNode->shape() != input->shape() ){
               backTensorNode = std::make_shared<Tensor<T>>(input->shape(),
               false , this );
            }
            backTensorNode->getData() = 1/2.f *(label->getData() - input->getData()) *(label->getData() - input->getData());
        }
        else{
            if(!backTensorNode || (backTensorNode->shape().size() != 1 && backTensorNode->shape()[0] != 1)){
               backTensorNode = std::make_shared<Tensor<T>>(std::initializer_list<int>{1 },false , this );
            }
            Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic> loss = 1/2.f *(label->getData() - input->getData()) *(label->getData() - input->getData()); 
            if(m_average){
                backTensorNode->getData() = loss.mean();
            }
            else{
                backTensorNode->getData() = loss.sum();
            }
        }
        return backTensorNode;

    } 

    void backward() override{
        preTensorNodes["input"]->getGrad() += -preTensorNodes["label"] ->getData() + preTensorNodes["input"] ->getData();
        if(m_average){
            preTensorNodes["input"]->getGrad() /= preTensorNodes["input"] ->getSize();
        }
    } 
};

template<class T>
class BCELoss:public CalculateNodeBase<T>{
private:
    bool m_average = true;
    bool m_reduce = true;

public:
    
    BCELoss(bool average = true , bool reduce = true):m_average(average) , m_reduce(reduce) {};

    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        if(data.size() != 2) return nullptr;
        auto input = *(data.begin());
        auto label = *(data.begin() + 1);
        input->addUseTime();
        label->addUseTime();
        preTensorNodes["input"] = input;
        preTensorNodes["label"] = label;
        if(!m_reduce){
            if(!backTensorNode || backTensorNode->shape() != input->shape() ){
               backTensorNode = std::make_shared<Tensor<T>>(input->shape(),
               false , this );
            }
            backTensorNode->getData() =-(label->getData()*input->getData().log() + (1 - label->getData())*(1 - input->getData()).log() ) ;
        }
        else{
            if(!backTensorNode || (backTensorNode->shape().size() != 1 && backTensorNode->shape()[0] != 1)){
               backTensorNode = std::make_shared<Tensor<T>>(std::initializer_list<int>{1 },false , this );
            }
            Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic> loss = -(label->getData()*input->getData().log() + (1 - label->getData())*(1 - input->getData()).log() ); 

            if(m_average){
                backTensorNode->getData() = loss.mean();
            }
            else{
                backTensorNode->getData() = loss.sum();
            }
        }
        return backTensorNode;

    } 

    void backward() override{
        preTensorNodes["input"]->getGrad() += - preTensorNodes["label"]->getData() / preTensorNodes["input"]->getData() + (1 - preTensorNodes["label"]->getData()) / (1 - preTensorNodes["input"]->getData());
        if(m_average){
            preTensorNodes["input"]->getGrad() /= preTensorNodes["input"]->getSize();
        }
    } 
};
#endif