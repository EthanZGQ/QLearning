#include "hip/hip_runtime.h"
#ifndef ACTIVATION_FUNCTION
#define ACTIVATION_FUNCTION

#include"CalculateNodeBase.cu"
#include"Tensor.cu"
#include<initializer_list>
#include<memory>


template<typename T>
struct sigmoidForward{
__device__ T operator()(T & nums){
    return 1.0f / (1.0f + __expf(-nums));
}
};

template<typename T>
struct sigmoidBackward{
__device__ T operator()(T & afterGrad , T & value){
   return afterGrad * (1 - value) * value;
}
};

template<class T>
class Sigmoid:public CalculateNodeBase<T>{
private:
    bool preCheck(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Need only one input !";
        auto input = * data.begin();
        preTensorNodes["input"] = input;
        input->addUseTime();

        if(!backTensorNode){
            backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
        }
        else{
            if(backTensorNode->shape() != input->shape()){
                backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
            }
        }
        return true;
    }
public:
    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        bool safe = preCheck(data);
        backTensorNode->getData() = 1.f/(1.f + preTensorNodes["input"]->getData().exp().cwiseInverse());
        return backTensorNode;
    }

    void backward() override{
        preTensorNodes["input"]->getGrad() += backTensorNode->getGrad() * (1.f - backTensorNode->getData()) * backTensorNode->getData();
    }

};



template<typename T>
struct reluForward{
__device__ T operator()(T & nums){
    T zero = static_cast<T>(0);
    return nums > zero ? nums : zero ;
}
};


template<typename T>
struct reluBackward{
__device__ T operator()(T & afterGrad , T & value){
    T zero = static_cast<T>(0);
    return value > zero ? afterGrad : zero ;
}
};

template<class T>
class ReLu:public CalculateNodeBase<T>{
private:
    bool preCheck(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Need only one input !";
        auto input = * data.begin();
        preTensorNodes["input"] = input;
        input->addUseTime();

        if(!backTensorNode){
            backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
        }
        else{
            if(backTensorNode->shape() != input->shape()){
                backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
            }
        }
        return true;
    }
public:
    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        bool safe = preCheck(data);
        auto fun = [](T data){return data > 0? data : 0 ;};
        backTensorNode->getData() = preTensorNodes["input"]->getData().unaryExpr(fun);
        return backTensorNode;
    }

    void backward() override{
        auto fun = [](T first , T second){ return second > 0 ? first : 0 ;};
        preTensorNodes["input"]->getGrad() += backTensorNode->getGrad().binaryExpr(preTensorNodes["input"]->getData() ,fun);
    }

};

<<<<<<< HEAD
template<typename T>
struct tanhForward{
__device__ T operator()(T & nums){
    T exp_1 = __expf(nums);
    T exp_2 = static_cast<T>(1.0f)/exp_1;
    return (exp_1 - exp_2)/(exp_1 + exp_2);
}
};

template<typename T>
struct tanhBackward{
__device__ T operator()(T & afterGrad , T & value){
    return afterGrad * (static_cast<T>(1.0f) - value * value);
}
};



template<class T>
class Tanh:public CalculateNodeBase<T>{
private:
    bool preCheck(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Need only one input !";
        auto input = * data.begin();
        preTensorNodes["input"] = input;
        input->addUseTime();

        if(!backTensorNode){
            backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
        }
        else{
            if(backTensorNode->shape() != input->shape()){
                backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
            }
        }
        return true;
    }
public:
    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        bool safe = preCheck(data);
        auto exp_1 = preTensorNodes["input"]->getData().exp();
        auto exp_2 = exp_1.cwiseInverse();
        backTensorNode->getData() = (exp_1 - exp_2) / (exp_1 + exp_2);
        return backTensorNode;
    }

    void backward() override{
        preTensorNodes["input"]->getGrad() += backTensorNode->getGrad() * (1 - backTensorNode->getData().pow(2));
    }

};




#endif