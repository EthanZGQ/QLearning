#include "hip/hip_runtime.h"
#ifndef ACTIVATION_FUNCTION
#define ACTIVATION_FUNCTION

#include"CalculateNodeBase.cu"
#include"Tensor.cu"
#include<initializer_list>
#include<memory>

template<class T>
class Sigmoid:public CalculateNodeBase<T>{
private:
    bool preCheck(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Need only one input !";
        auto input = * data.begin();
        preTensorNodes["input"] = input;
        input->addUseTime();

        if(!backTensorNode){
            backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
        }
        else{
            if(backTensorNode->shape() != input->shape()){
                backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
            }
        }
        return true;
    }
public:
    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        bool safe = preCheck(data);
        backTensorNode->getData() = 1.f/(1.f + preTensorNodes["input"]->getData().exp().cwiseInverse());
        return backTensorNode;
    }

    void backward() override{
        preTensorNodes["input"]->getGrad() += backTensorNode->getGrad() * (1.f - backTensorNode->getData()) * backTensorNode->getData();
    }

};



template<class T>
class ReLu:public CalculateNodeBase<T>{
private:
    bool preCheck(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Need only one input !";
        auto input = * data.begin();
        preTensorNodes["input"] = input;
        input->addUseTime();

        if(!backTensorNode){
            backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
        }
        else{
            if(backTensorNode->shape() != input->shape()){
                backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
            }
        }
        return true;
    }
public:
    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        bool safe = preCheck(data);
        auto fun = [](T data){return data > 0? data : 0 ;};
        backTensorNode->getData() = preTensorNodes["input"]->getData().unaryExpr(fun);
        return backTensorNode;
    }

    void backward() override{
        auto fun = [](T first , T second){ return second > 0 ? first : 0 ;};
        preTensorNodes["input"]->getGrad() += backTensorNode->getGrad().binaryExpr(preTensorNodes["input"]->getData() ,fun);
    }

};

template<class T>
class Tanh:public CalculateNodeBase<T>{
private:
    bool preCheck(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Need only one input !";
        auto input = * data.begin();
        preTensorNodes["input"] = input;
        input->addUseTime();

        if(!backTensorNode){
            backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
        }
        else{
            if(backTensorNode->shape() != input->shape()){
                backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
            }
        }
        return true;
    }
public:
    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        bool safe = preCheck(data);
        auto exp_1 = preTensorNodes["input"]->getData().exp();
        auto exp_2 = exp_1.cwiseInverse();
        backTensorNode->getData() = (exp_1 - exp_2) / (exp_1 + exp_2);
        return backTensorNode;
    }

    void backward() override{
        preTensorNodes["input"]->getGrad() += backTensorNode->getGrad() * (1 - backTensorNode->getData().pow(2));
    }

};





#endif