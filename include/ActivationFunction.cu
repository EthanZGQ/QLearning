#include "hip/hip_runtime.h"
#ifndef ACTIVATION_FUNCTION
#define ACTIVATION_FUNCTION

#include"CalculateNodeBase.cu"
#include"Tensor.cu"
#include<initializer_list>
#include<memory>

template<class T>
class Sigmoid:public CalculateNodeBase<T>{
private:
    bool preCheck(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Need only one input !";
        auto input = * data.begin();
        preTensorNodes["input"] = input;
        input->addUseTime();

        if(!backTensorNode){
            backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
        }
        else{
            if(backTensorNode->shape() != input->shape()){
                backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
            }
        }
        return true;
    }
public:
    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        bool safe = preCheck(data);
        backTensorNode->getData() = 1.f/(1.f + preTensorNodes["input"]->getData().exp().cwiseInverse());
        return backTensorNode;
    }

    void backward() override{
        preTensorNodes["input"]->getGrad() = (1.f - backTensorNode->getData()) * backTensorNode->getData();
    }

};



template<class T>
class ReLu:public CalculateNodeBase<T>{
private:
    bool preCheck(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Need only one input !";
        auto input = * data.begin();
        preTensorNodes["input"] = input;
        input->addUseTime();

        if(!backTensorNode){
            backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
        }
        else{
            if(backTensorNode->shape() != input->shape()){
                backTensorNode = std::make_shared<Tensor<T>>(input->shape() , false , this);
            }
        }
        return true;
    }
public:
    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        bool safe = preCheck(data);
        auto fun = [](T data){return data > 0? data : 0 ;};
        backTensorNode->getData() = preTensorNodes["input"]->getData().unaryExpr(fun);
        return backTensorNode;
    }

    void backward() override{
        auto fun = [](T first , T second){ return second > 0 ? first : 0 ;};
        preTensorNodes["input"]->getGrad() = backTensorNode->getGrad().binaryExpr(preTensorNodes["input"]->getData() ,fun);
    }

};


#endif