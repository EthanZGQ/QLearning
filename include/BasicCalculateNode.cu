#include "hip/hip_runtime.h"
#ifndef BASIC_CALCULATE_NODE
#define BASIC_CALCULATE_NODE

#include"CalculateNodeBase.cu"
#include<Tensor.cu>
#include<memory>
#include<initializer_list>


template<class T>
class Linear:public CalculateNodeBase<T>{
private:
    int m_inFeature;
    int m_outFeature;
    bool m_bias = true;

    bool inferShape(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Only need one input !";
        auto input = *(data.begin());
        if(input->shape().back() != m_inFeature) throw "The input last dim should same as the inFeature !";
        preTensorNodes["input"] = input;
        input->addUseTime();
        if(!backTensorNode){
            std::vector<int> tempSize = input->shape();
            tempSize.back() = m_outFeature;
            backTensorNode = std::make_shared<Tensor<T>>(tempSize , false , this);
        }
        else{
            std::vector<int> tempSize = input->shape();
            tempSize.back() = m_outFeature;
            if(tempSize != backTensorNode->shape()){
                backTensorNode = std::make_shared<Tensor<T>>(tempSize , false , this);
            }
        }
        return true;
    }

    void compute(){
        backTensorNode->getData() = preTensorNodes["weights"]->getData().matrix() * (preTensorNodes["input"]->getData()).matrix();
        if(m_bias){
            backTensorNode->getData().colwise() += preTensorNodes["bias"]->getData().rowwise().sum();
        }
    }


public:
    Linear(int in_feature , int out_feature , bool bias = true):m_inFeature(in_feature) , m_outFeature(out_feature) , m_bias(bias) {
        preTensorNodes["weights"] = std::make_shared<Tensor<T>>(std::initializer_list<int>({in_feature , out_feature}) , true);
        if(m_bias){
            preTensorNodes["bias"] = std::make_shared<Tensor<T>>(std::initializer_list<int>({1 , out_feature}) , true);
        }
    };

    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        bool checkOk = inferShape(data);
        compute();
        return backTensorNode;
    }

    void backward() override{
        preTensorNodes["weights"]->getGrad() = preTensorNodes["weights"]->getGrad().matrix() +  (backTensorNode->getGrad()).matrix() * (preTensorNodes["input"]->getData().transpose()).matrix();
        preTensorNodes["input"]->getGrad() = preTensorNodes["input"]->getGrad().matrix() +  (preTensorNodes["weights"]->getData().transpose()).matrix() * backTensorNode->getGrad().matrix();
        if(m_bias){
            preTensorNodes["bias"]->getGrad() = preTensorNodes["bias"]->getGrad()  +  backTensorNode->getGrad().rowwise().sum();
        }
    }

};


template<class T>
class Conv2d :public CalculateNodeBase<T>{
public:
    int m_inChannals;
    int m_outChannals;
    int m_kernalSize;
    int m_stride;
    int m_padding;
    int m_dilation;
    std::shared_ptr<Tensor<T>> m_img2colData = nullptr;

    bool inferShape(std::initializer_list<std::shared_ptr<Tensor<T>>> & data){
        if(data.size() != 1) throw "Only need one input !";
        auto input = *(data.begin());
        if(input->shape().size() != 4) throw "The conv2d need shape 4 dim such as (batch,channal,height,width)";
        if(input->shape()[1]!= m_inChannals ) throw "The input second dim should same as the inChannals !";
        preTensorNodes["input"] = input;
        input->addUseTime();
        prepareImg2colData();
    }

    void prepareImg2colData(){
        auto inputShape = preTensorNodes["input"]->shape();
        int batch = inputShape.front();
        int height = inputShape[2];
        int width = inputShape.back();

        int colTime = std::ceilf((width + 2 * m_padding - m_kernalSize - (m_kernalSize - 1) *m_dilation + 1)
        /static_cast<float>(m_stride)); // 每行进行了几次卷积
        int rowTime = std::ceilf((height + 2 * m_padding - m_kernalSize - (m_kernalSize - 1) *m_dilation + 1)
        /static_cast<float>(m_stride)); // 每行进行了几次卷积
        int matrixRow = m_kernalSize * m_kernalSize * m_inChannals;
        int matrixCol = colTime * rowTime * batch;
        if(m_img2colData == nullptr){
            m_img2colData = std::make_shared<Tensor<T>>(std::initializer_list<int>({matrixCol , matrixRow}));
            backTensorNode = std::make_shared<Tensor<T>>(std::initializer_list<int>({batch , m_outChannals , rowTime , colTime}) , false , this);
        }
        else{
            std::vector<int> nowShape = {batch , m_outChannals , rowTime , colTime};
            if(nowShape != backTensorNode->shape()){
                m_img2colData = std::make_shared<Tensor<T>>(std::initializer_list<int>({matrixCol , matrixRow}));
                backTensorNode = std::make_shared<Tensor<T>>(nowShape , false , this);
            }
        }
    }


    void compute(){
        img2colCpu();
        Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic> value = preTensorNodes["weights"]->getData().matrix() * m_img2colData->getData().matrix();
        std::cout << "the output data is " << std::endl << value << std::endl << std::endl;
        int batch = backTensorNode->shape().front();
        int lineLen = value.size() / (m_outChannals * batch);
        int colLen = backTensorNode->shape().back();
        int rowLen = backTensorNode->shape()[2];
        for(int tempBatch = 0 ; tempBatch < batch ; ++tempBatch){
            Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic> tempValue = value.block(0 , lineLen * tempBatch , m_outChannals , lineLen);
            tempValue.transposeInPlace();
            tempValue.resize(colLen , rowLen*m_outChannals);
            backTensorNode->getData().block(0 , tempBatch * m_outChannals * rowLen , colLen , m_outChannals * rowLen) = tempValue;
        }

    }

    void backwardCompute(){
        int batch = backTensorNode->shape().front();
        int colLen = backTensorNode->shape().back();
        int rowLen = backTensorNode->shape()[2];
        int lineLen = colLen * rowLen;
        Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic> backNodeGrad(m_outChannals , batch * lineLen);
        for(int tempBatch = 0 ; tempBatch < batch ; ++tempBatch){
            Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic> tempValue = backTensorNode->getGrad().block(0 , m_outChannals * rowLen* tempBatch , colLen , m_outChannals * rowLen);
            tempValue.resize(lineLen , m_outChannals);
            tempValue.transposeInPlace();
            backNodeGrad.block(0 , tempBatch * lineLen , m_outChannals , lineLen) = tempValue;
        }
        preTensorNodes["weights"]->getGrad() += (backNodeGrad.matrix() * m_img2colData->getData().transpose().matrix()).array();
        m_img2colData->getGrad() += (preTensorNodes["weights"]->getData().transpose().matrix() * backNodeGrad.matrix()).array();
        img2colbackwardCpu();
        m_img2colData->getGrad().setZero();
    }

    void img2colbackwardCpu(){
        auto inputShape = preTensorNodes["input"]->shape();
        int batch = inputShape[0] , height = inputShape[2] , width = inputShape[3];
        int colTime = backTensorNode->shape().back() , rowTime = backTensorNode->shape()[2];
        int imgSize = width * height; //一个图片的大小
        int featureMapSize = m_inChannals * imgSize; //一个特征图的大小
        int flatKernalSize = m_kernalSize * m_kernalSize; //一个单层卷积核的大小
        int oneLineSize = flatKernalSize * m_inChannals; //img2col之后 一列的长度
        int oneLayerSize = oneLineSize* colTime * rowTime; //一个特征图 img2col之后的内存大小
        int realKernalSize = m_kernalSize + (m_kernalSize - 1) * m_dilation; //经过稀疏卷积后的卷积和的宽高长度
        T * output = preTensorNodes["input"]->getGrad().data();
        T * input = m_img2colData->getGrad().data();
        for(int _batch = 0 ; _batch < batch ; ++_batch){
            for(int row = -m_padding ; row <= height + m_padding - realKernalSize; row += m_stride){
                for(int col = -m_padding ; col <= width + m_padding - realKernalSize; col += m_stride){
                    for(int _feature = 0 ; _feature < m_inChannals ; ++ _feature){
                        for(int y = 0 ; y < m_kernalSize ; ++y){
                            for(int x = 0 ; x < m_kernalSize ; ++x){
                                T value ;
                                int realY = y*(m_dilation + 1);
                                int realX = x*(m_dilation + 1);
                                if(realY + row < 0 || realY + row >= height || realX + col < 0 || realX + col >= width){ //判断边界关系
                                    continue;
                                } 
                                else {
                                    int index = _batch * oneLayerSize + ((row + m_padding)/m_stride) * colTime *oneLineSize + 
                                    (col+m_padding)/m_stride * oneLineSize + flatKernalSize * _feature + y*m_kernalSize + x ;
                                    value = input[index]; 
                                }                //一个特征图的内存长度        //一行之后的内存长度       //一行之中的内存长度  
                                int imgIndex = _batch * featureMapSize + _feature * imgSize + (realY + row)*width + (realX + col);
                                output[imgIndex] += value;
                            }
                        }
                    }
                }
            }
        }
    }

    void img2colCpu(){
        auto inputShape = preTensorNodes["input"]->shape();
        int batch = inputShape[0] , height = inputShape[2] , width = inputShape[3];
        int colTime = backTensorNode->shape().back() , rowTime = backTensorNode->shape()[2];
        int imgSize = width * height; //一个图片的大小
        int featureMapSize = m_inChannals * imgSize; //一个特征图的大小
        int flatKernalSize = m_kernalSize * m_kernalSize; //一个单层卷积核的大小
        int oneLineSize = flatKernalSize * m_inChannals; //img2col之后 一列的长度
        int oneLayerSize = oneLineSize* colTime * rowTime; //一个特征图 img2col之后的内存大小
        int realKernalSize = m_kernalSize + (m_kernalSize - 1) * m_dilation; //经过稀疏卷积后的卷积和的宽高长度
        T * input = preTensorNodes["input"]->getData().data();
        T * output = m_img2colData->getData().data();
        for(int _batch = 0 ; _batch < batch ; ++_batch){
            for(int row = -m_padding ; row <= height + m_padding - realKernalSize; row += m_stride){
                for(int col = -m_padding ; col <= width + m_padding - realKernalSize; col += m_stride){
                    for(int _feature = 0 ; _feature < m_inChannals ; ++ _feature){
                        for(int y = 0 ; y < m_kernalSize ; ++y){
                            for(int x = 0 ; x < m_kernalSize ; ++x){
                                T value ;
                                int realY = y*(m_dilation + 1);
                                int realX = x*(m_dilation + 1);
                                if(realY + row < 0 || realY + row >= height || realX + col < 0 || realX + col >= width){ //判断边界关系
                                    value = 0;
                                } 
                                else {
                                    int imgIndex = _batch * featureMapSize + _feature * imgSize + (realY + row)*width + (realX + col);
                                    value = input[imgIndex];
                                }                //一个特征图的内存长度        //一行之后的内存长度       //一行之中的内存长度  
                                int index = _batch * oneLayerSize + ((row + m_padding)/m_stride) * colTime *oneLineSize + 
                                (col+m_padding)/m_stride * oneLineSize + flatKernalSize * _feature + y*m_kernalSize + x ;
                                output[index] = value; 
                            }
                        }
                    }
                }
            }
        }
    }

// public:
    Conv2d(int inChannals , int outChannals , int kernalSize , int padding = 0 , int stride = 1 , int dilation = 0):
    m_inChannals(inChannals) , m_outChannals(outChannals) , m_kernalSize(kernalSize) , m_padding(padding) , m_stride(stride) , m_dilation(dilation){
        preTensorNodes["weights"] = std::make_shared<Tensor<T>>(std::initializer_list<int>({ m_inChannals, m_kernalSize , m_kernalSize , m_outChannals}) , true );
        preTensorNodes["weights"]->getData().setConstant(1);
    }
    std::shared_ptr<Tensor<T>> forward(std::initializer_list<std::shared_ptr<Tensor<T>>> data) override{
        inferShape(data);
        compute(); 
        return backTensorNode;
    }

    void backward(){
        backwardCompute();
    };

};


#endif //BASIC_CALCULATE_NODE