#ifndef TENSOR
#define TENSOR
#include"CalculateNodeBase.cu"
#include<Eigen\Dense>
#include<vector>
#include<iostream>
#include<queue>
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"hipblas.h"



template<class T>
class Tensor{
private:
    Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic> m_cpuData;
    Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic> m_cpuGrad;
    T * m_gpuData = nullptr;
    T * m_gpuGrad = nullptr;
    std::vector<int> m_shape;
    bool m_needGrad = false;
    bool m_inCuda;
    CalculateNodeBase<T> *m_preCalculateNode = nullptr;
    int m_useTime = 0;
    int m_size = 0;

public:
    
    Tensor(std::vector<int> size , bool needGrad = false ,CalculateNodeBase<T> *fatherNode = nullptr , bool useCuda = false){
        m_shape = size;
        int rows = 1;
        for(auto & x : m_shape){
            if(!x) throw "The size number should greater than 0 !";
            rows *= x;
        }
        m_size = rows;
        rows /= m_shape.back();
        m_needGrad = needGrad;
        m_inCuda = useCuda;
        if(typeid(T) != typeid(float)  && typeid(T) != typeid(int) && typeid(T) != typeid(double)){
            throw "only support float int double ! I'm so sorry !";
        }
        if(!m_inCuda){
            m_cpuData = Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic>::Random(m_shape.back() , rows);
            if(m_needGrad)
            m_cpuGrad = Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic>::Zero(m_shape.back() , rows);
        }
        else{
            auto cudaInfo = hipMalloc((void **) & m_gpuData , sizeof(T) * m_size);
            if(cudaInfo != hipSuccess){
                throw "can't hipMalloc the gpuData !";
            } 
            if(m_needGrad){
                cudaInfo = hipMalloc((void **) & m_gpuGrad , sizeof(T) * m_size);
                if(cudaInfo != hipSuccess){
                    throw "can't hipMalloc the gpuGrad !";
                } 
            }
        }
        if(m_gpuData) std::cout << "get the gpu data mem !";
        if(m_gpuGrad) std::cout << "get the gpu grad mem !";
        m_preCalculateNode = fatherNode;
    }

    Tensor(std::initializer_list<int> size , Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic> & data , 
    bool needGrad = false , CalculateNodeBase<T> * fatherNode = nullptr){
        m_shape = size ;
        if(data.rows() != m_shape.back()){
            throw "input data last dim should same sa the size last dim !";
        }
        int tempSize = 1;
        for(auto x : m_shape){
            tempSize *= x;
            if(!x) throw "The size number should greater than 0 !";
        }
        if(data.cols() * data.rows() != tempSize){
            throw "input data size should same as input size";
        }
        m_size = tempSize;
        m_needGrad = needGrad;
        m_cpuData = data;
        if(m_needGrad){
            m_cpuGrad = Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic>::Zero(data.rows() , data.cols());
        }
        m_preCalculateNode = fatherNode;
    }

    void cuda(){
        if(m_inCuda) return;
        m_inCuda = true;
        hipError_t cudaInfo;
        if(m_gpuData == nullptr){
            auto cudaInfo = hipMalloc((void **) & m_gpuData , sizeof(T) * m_size);
            if(cudaInfo != hipSuccess){
                std::cout << "to cuda ,hipMalloc m_gpuData failed !";
                throw  "to cuda ,hipMalloc m_gpuData failed !";
            }
        }

        cudaInfo = hipMemcpy(m_gpuData , m_cpuData.data() , m_size * sizeof(T) , hipMemcpyHostToDevice);
        if(cudaInfo != hipSuccess){
            std::cout << "to cuda ,hipMemcpy m_gpuData failed !";
            throw  "to cuda ,hipMemcpy m_gpuData failed !";
        }

        if(m_needGrad){
            if(m_gpuGrad == nullptr){
                auto cudaInfo = hipMalloc((void **) & m_gpuGrad , sizeof(T) * m_size);
                if(cudaInfo != hipSuccess){
                    std::cout << "to cuda ,hipMalloc m_gpuGrad failed !";
                    throw  "to cuda ,hipMalloc m_gpuGrad failed !";
                }
            }
            cudaInfo = hipMemcpy(m_gpuGrad , m_cpuGrad.data() , m_size * sizeof(T) , hipMemcpyHostToDevice);
            if(cudaInfo != hipSuccess){
                std::cout << "to cuda ,hipMemcpy m_gpuGrad failed !";
                throw  "to cuda ,hipMemcpy m_gpuGrad failed !";
            }
        }
    }

    void host(){
        if(!m_inCuda) return ;
        if(m_cpuData.rows() == 0){
            m_cpuData = Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic>(m_shape.back() , m_size / m_shape.back());
        }
        if(m_needGrad && m_cpuGrad.rows() == 0){
            m_cpuGrad = Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic>(m_shape.back() , m_size / m_shape.back());
        }
        auto cudaInfo = hipMemcpy(m_cpuData.data() ,m_gpuData , m_size * sizeof(T) , hipMemcpyDeviceToHost);
        if(cudaInfo != hipSuccess){
            std::cout << "to host ,hipMemcpy m_gpuData failed !";
            throw  "to host ,hipMemcpy m_gpuData failed !";
        }
        if(m_gpuGrad){
            cudaInfo = hipMemcpy(m_cpuGrad.data() , m_gpuGrad , m_size * sizeof(T) , hipMemcpyHostToDevice);
            if(cudaInfo != hipSuccess){
                std::cout << "to host ,hipMemcpy m_gpuGrad failed !";
                throw  "to host ,hipMemcpy m_gpuGrad failed !";
            }
        }
        m_inCuda = false;
    }

    Eigen::Array<T,Eigen::Dynamic , Eigen::Dynamic> & getData(){
        if(m_inCuda){
            if(m_cpuData.rows() == 0){
                m_cpuData = Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic>(m_shape.back() , m_size / m_shape.back());
            }
            auto cudaInfo = hipMemcpy(m_cpuData.data() ,m_gpuData , m_size * sizeof(T) , hipMemcpyDeviceToHost);
            if(cudaInfo != hipSuccess){
                std::cout << "to host ,hipMemcpy m_gpuData failed !";
                throw  "to host ,hipMemcpy m_gpuData failed !";
            }
        }
        return m_cpuData;
    }

    Eigen::Array<T,Eigen::Dynamic , Eigen::Dynamic> & getGrad(){
        if(m_inCuda){
            if(m_cpuGrad.rows() == 0){
                m_cpuGrad = Eigen::Array<T , Eigen::Dynamic , Eigen::Dynamic>(m_shape.back() , m_size / m_shape.back());
            }
            auto cudaInfo = hipMemcpy(m_cpuGrad.data() ,m_gpuGrad , m_size * sizeof(T) , hipMemcpyDeviceToHost);
            if(cudaInfo != hipSuccess){
                std::cout << "to host ,hipMemcpy m_gpuGrad failed !";
                throw  "to host ,hipMemcpy m_gpuGrad failed !";
            }
        }
        return m_cpuGrad;
    }


    void backward(){
        std::queue<CalculateNodeBase<T>*> myQ;
        myQ.push(m_preCalculateNode);
        while(myQ.size()){
            auto calNode = myQ.front();
            myQ.pop();
            calNode->backward();
            calNode->backTensorNode->zeroGrad();
            for(auto & x : calNode->preTensorNodes){
                auto ptr = x.second;
                ptr->subUseTime();
                auto tempPtr = ptr->getPreCalculateNode();
                if( ptr->useTimeEmpty() && tempPtr) myQ.push(tempPtr);
            }
        }
    }
 
    ~Tensor(){
        if(m_gpuData) {
            std::cout << "delete data  okkkk ";
            hipFree(m_gpuData);
            }
        if(m_gpuGrad){
            std::cout << "delete grad  okkkk ";
            hipFree(m_gpuGrad);
        } 
    }

    T * getCudaDataPtr(){
        return m_gpuData;
    }

    T * getCudaGradPtr(){
        return m_gpuGrad;
    }

    bool inCuda(){
        return m_inCuda;
    }

    void adjust(T lr){
        m_cpuData -= m_cpuGrad * lr;
    }

    void zeroGrad(){
        m_cpuGrad.setZero();
    }

    bool useTimeEmpty(){
        return m_useTime == 0;
    }

    void addUseTime(){
        ++m_useTime;
    }

    void subUseTime(){
        --m_useTime;
    }

    int getUseTime(){
        return m_useTime;
    }
    int getSize(){
        return m_size;
    }

    CalculateNodeBase<T> * getPreCalculateNode(){ 
        return m_preCalculateNode;
    }

    const std::vector<int> & shape(){
        return m_shape;
    }

    bool needGrad(){
        return m_needGrad;
    }

};


#endif