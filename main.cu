#include<iostream>
#include<Eigen\Dense>
#include"Tensor.cu"
#include<string>
#include<exception>
#include"BasicCalculateNode.cu"
#include<memory>
#include"ActivationFunction.cu"
#include"Module.cu"
#include"Optimizer.cu"
#include"LossFunction.cu"
#include<chrono>
#include"DatasetBase.cu"
#include"DataLoader.cu"
#include"hipblas.h"

// class myHandle{
// public:
//     int a;
//     static hipblasHandle_t cublasH ;
// };

// hipblasHandle_t myHandle::cublasH = nullptr;

// class InitHandle{
// public:
//     InitHandle(){
//         auto cudaInfo = hipblasCreate(&myHandle::cublasH);
//         if(cudaInfo != HIPBLAS_STATUS_SUCCESS){
//             std::cout << "can't get the cublas handle !" << std::endl;
//         }
//         else{
//             std::cout << "init handle okkkkk!";
//         }
//     }

//     ~InitHandle(){
//         auto cudaInfo = hipblasDestroy(myHandle::cublasH);
//         if(cudaInfo != HIPBLAS_STATUS_SUCCESS){
//             std::cout << "can't free the cublas handle !" << std::endl;
//         }
//         else{
//             std::cout << "free handle okkkkk!";
//         }
//     }
// };

// InitHandle init;
float mul(int left , int right )
{
    return -left * right;
}

int main(){
    int a = 2 , b = -1 ,c = 3 ,d = 1;
    float e,f;
    e = (a *c - mul(b,d))/(c*c - mul(d,d));
    f = (b * c - a * d)/(c*c - mul(d,d));
    std::cout << e << " " << f;
    return 0;
}